#include "hip/hip_runtime.h"



#include <vector>
#include <algorithm>
#include <numeric>
#include <iostream>
#include <chrono>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void BinaryReductionStep(int ** sections, const size_t increment, const size_t width)
{
    int * sectionA = sections[blockIdx.x];
    int * sectionB = sections[blockIdx.x + increment];

    for (int i=threadIdx.x; i<width; i+=blockDim.x)
    {
        sectionA[i] += sectionB[i];
    }
}

__global__ void RotatingReductionStep(int ** sections, const size_t increment, int * output)
{
    size_t srcSectionIndex = (blockIdx.x + increment) % gridDim.x;
    int * inputSection = sections[srcSectionIndex];

    size_t index = blockDim.x * blockIdx.x + threadIdx.x;

    output[index] += inputSection[index];
}

std::vector<int*> createDevData(std::vector<int>& srcData, const size_t sections)
{
    std::vector<int*> dev_data(sections);

    for (int i = 0; i < sections; i++)
    {
        gpuErrchk( hipMalloc((void**)&dev_data[i], srcData.size() * sizeof(int)));
        gpuErrchk( hipMemcpy(dev_data[i], srcData.data(), srcData.size() * sizeof(int), hipMemcpyDefault));
    }

    return dev_data;
}

int** getDevDataSections(std::vector<int*> src)
{
    int ** dev_data;
    gpuErrchk( hipMalloc((void**)&dev_data, src.size() * sizeof(int*)));
    gpuErrchk( hipMemcpy(dev_data, src.data(), src.size() * sizeof(int*),hipMemcpyDefault));

    return dev_data;
}

int main()
{
    const size_t N = 8192;

    const size_t sections = 16;

    const size_t blockwidth = 512;



    std::vector<int> srcData(N);
    std::iota(srcData.begin(), srcData.end(),1);

    hipStream_t stream;
    hipStreamCreate(&stream);

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    auto dataB = createDevData(srcData, sections);
    auto dataB_sections = getDevDataSections(dataB);
    
    std::vector<int> resultB(N);
    float msB;
    {
        hipEventRecord(start, stream);

        BinaryReductionStep<<<8, blockwidth,0, stream>>>(dataB_sections, 8, N);
        BinaryReductionStep<<<4, blockwidth,0, stream>>>(dataB_sections, 4, N);
        BinaryReductionStep<<<2, blockwidth,0, stream>>>(dataB_sections, 2, N);
        BinaryReductionStep<<<1, blockwidth,0, stream>>>(dataB_sections, 1, N);

        hipEventRecord(end, stream);

        hipEventSynchronize(end);

        hipEventElapsedTime(&msB, start, end);

        hipMemcpy(resultB.data(), dataB[0], srcData.size() * sizeof(int), hipMemcpyDefault);
    }

    auto dataR = createDevData(srcData, sections);
    auto dataR_sections = getDevDataSections(dataR);

    std::vector<int> resultR(N);
    float msR;
    {
        int * dev_RO;
        hipMalloc((void**)&dev_RO, srcData.size() * sizeof(int));
        hipMemset(dev_RO, 0, srcData.size() * sizeof(int));

        hipEventRecord(start, stream);

        for (int i= 0; i < sections; i++)
        {
            RotatingReductionStep<<<16, blockwidth,0, stream>>>(dataR_sections, i, dev_RO);
        }    

        hipEventRecord(end, stream);

        hipEventSynchronize(end);

        hipEventElapsedTime(&msR, start, end);

        hipMemcpy(resultR.data(), dev_RO, srcData.size() * sizeof(int), hipMemcpyDefault);
    }

    if ( std::equal(resultR.begin(), resultR.end(), resultB.begin(), resultB.end() ) )
    {
        std::cout<<"Binary: "<<msR<<" - Rotating: "<<msB<<"\n";
    }
    else
    {
        std::cout<<"Fail\n";
        for (int i = 0; i < 12; i++)
        {
            std::cout<<"B"<<resultB.at(i)<<" - R"<<resultR.at(i)<<"\n";
        }
    }


    return 0;
}

